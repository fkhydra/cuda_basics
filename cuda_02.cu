#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
#include <stdio.h>

int main()
{
 int devcount;
 int dev, driverVersion = 0, runtimeVersion = 0;

 hipGetDeviceCount(&devcount);
 if (devcount == 0) printf("No supported CUDA device found!\n");
 else printf("%i CUDA device(s) found...", devcount);

 for (dev = 0; dev < devcount; ++dev)
 {
  hipSetDevice(dev);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, dev);
  printf("\n%d. device name: \"%s\"\n", dev, deviceProp.name);

  hipDriverGetVersion(&driverVersion);
  hipRuntimeGetVersion(&runtimeVersion);

  printf("  CUDA driver version / Runtime version          %d.%d / %d.%d\n",
   driverVersion / 1000, (driverVersion % 100) / 10,
   runtimeVersion / 1000, (runtimeVersion % 100) / 10);

  printf("  CUDA Capability version:   %d.%d\n",
   deviceProp.major, deviceProp.minor);

  char msg[256];
  sprintf_s(msg, sizeof(msg),
   "  Total VRAM:     %.0f MBytes "
   "(%llu bytes)\n",
   static_cast<float>(deviceProp.totalGlobalMem / 1048576.0f),
   (unsigned long long)deviceProp.totalGlobalMem);
  printf("%s", msg);

  //_ConvertSMVer2Cores needs cuda-samples to be installed
  printf("  (%2d) Multiprocessor, (%3d) CUDA cores/MP: %d CUDA cores\n",
   deviceProp.multiProcessorCount,
   _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
   _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) *
   deviceProp.multiProcessorCount);

  printf(
   "  GPU max. clock:                            %.0f MHz (%0.2f "
   "GHz)\n",
   deviceProp.clockRate * 1e-3f, deviceProp.clockRate * 1e-6f);

  printf("  VRAM clock:                             %.0f Mhz\n",
   deviceProp.memoryClockRate * 1e-3f);
  printf("  VRAM transfer rate:                  %d-bit\n",
   deviceProp.memoryBusWidth);
  printf("  Warp size:                                %d\n",
   deviceProp.warpSize);
  printf("  Max. threads / multiprocessor: %d\n",
   deviceProp.maxThreadsPerMultiProcessor);
  printf("  Max. threads / block:  %d\n",
   deviceProp.maxThreadsPerBlock);
  printf("  Max. block dimensions (x,y,z):  (%d, %d, %d)\n",
   deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1],
   deviceProp.maxThreadsDim[2]);
  printf("  Max. Grid dimensions (x,y,z): (%d, %d, %d)\n",
   deviceProp.maxGridSize[0], deviceProp.maxGridSize[1],
   deviceProp.maxGridSize[2]);
 }
 return 0;
}
