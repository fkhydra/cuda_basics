#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void GPU_kernel(int max_itemcount)
{
 int i;
 int startindex = threadIdx.x;
 int step = blockDim.x;
 for (i = startindex; i < max_itemcount; i += step)
 {
  printf("%i\n", i);
 }
}

int main(void)
{
 GPU_kernel << < 1, 10 >> > (100);
 hipDeviceSynchronize();
 printf("Finished execution!\n");
 return 0;
}
