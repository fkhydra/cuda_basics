#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void GPU_kernel(int max_itemcount)
{
 int i;
 for (i = 0; i < max_itemcount; ++i)
 {
  printf("%i\n", i);
 }
}

int main(void)
{
 GPU_kernel << < 1, 1 >> > (100);
 hipDeviceSynchronize();
 printf("Finished execution!\n");
 return 0;
}
