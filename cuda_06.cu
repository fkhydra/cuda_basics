#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void GPU_kernel(int max_itemcount)
{
 int i;
 int startindex = threadIdx.x + (blockIdx.x * blockDim.x);
 int step = blockDim.x * gridDim.x;
 for (i = startindex; i < max_itemcount; i += step)
 {
  printf("%i\n", i);
 }
}

int main(void)
{
 int thread_count = 128;
 int block_count = (100000 + thread_count - 1) / thread_count;

 GPU_kernel <<< block_count, thread_count >>> (100000);
 hipDeviceSynchronize();
 printf("Finished execution!\n");
 return 0;
}
