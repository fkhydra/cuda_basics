#include "hip/hip_runtime.h"

#include <stdio.h>

int main()
{
 int devcount;
 hipGetDeviceCount(&devcount);
 printf("%i device(s) found...", devcount);
 return 0;
}
