#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void set_value(int *c, int number)
{
    c[0] = number;
}

int main()
{
 int *dev0_data;
 int *dev1_data;
 int current_data=0;

 hipSetDevice(0);
 hipMalloc((void**)&dev0_data, 1 * sizeof(int));
 hipMemcpy(dev0_data, &current_data, 1 * sizeof(int), hipMemcpyHostToDevice); 
 hipSetDevice(1);
 hipMalloc((void**)&dev1_data, 1 * sizeof(int)); 
 hipMemcpy(dev0_data, &current_data, 1 * sizeof(int), hipMemcpyHostToDevice);
 printf("Memoriy allocated...\n");

 hipSetDevice(0);
 set_value << <1, 1 >> > (dev0_data,1);
 hipDeviceSynchronize();
 hipSetDevice(1);
 set_value << <1, 1 >> > (dev1_data, 9);
 hipDeviceSynchronize();
 printf("Kernels… ok\n");

 hipSetDevice(0);
 hipMemcpy(&current_data, dev0_data, 1 * sizeof(int), hipMemcpyDeviceToHost);
 printf("DEV0: %i\n", current_data);
 hipSetDevice(1);
 hipMemcpy(&current_data, dev1_data, 1 * sizeof(int), hipMemcpyDeviceToHost);
 printf("DEV1: %i\n", current_data);

 hipMemcpyPeer(dev0_data,0, dev1_data,1,1*sizeof(int));
 printf("Swap… ok\n");

 hipSetDevice(0);
 hipMemcpy(&current_data, dev0_data, 1 * sizeof(int), hipMemcpyDeviceToHost);
 printf("DEV0: %i\n",current_data);
 hipSetDevice(1);
 hipMemcpy(&current_data, dev1_data, 1 * sizeof(int), hipMemcpyDeviceToHost);
 printf("DEV1: %i\n", current_data);
 
 hipFree(dev0_data);
 hipFree(dev1_data);
 return 0;
}
