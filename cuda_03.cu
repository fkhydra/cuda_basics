#include "hip/hip_runtime.h"

#include <stdio.h>

void CPU_kernel(int max_itemcount)
{
 int i;
 for (i = 0; i < max_itemcount; ++i)
 {
  printf("%i\n", i);
 }
}

int main(void)
{
 CPU_kernel(100);
 printf("Finished execution!\n");
 return 0;
}
