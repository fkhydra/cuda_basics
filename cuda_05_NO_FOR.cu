#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void GPU_kernel(int max_itemcount)
{
 int current_index = threadIdx.x + (blockIdx.x * blockDim.x);
 if (current_index < max_itemcount) printf("%i\n", current_index);
}

int main(void)
{
 GPU_kernel << < 1, 10 >> > (100);
 hipDeviceSynchronize();
 printf("Finished execution!\n");
 return 0;
}
